#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<hip/hip_runtime.h>
#define row1 2
#define col1 3
#define row2 3
#define col2 2
__global__ void matrixproduct(int *l,int *m, int *n)
{
int x=blockIdx.x;
int y=blockIdx.y;
__shared__ int p[col1];
int i;
int k=threadIdx.x;
n[col2*y+x]=0;
p[k]=l[col1*y+k]*m[col2*k+x];
__syncthreads();
for(i=0;i<col1;i++)
n[col2*y+x]=n[col2*y+x]+p[i];
}
int main()
{
int a[row1][col1]={1,1,3,3,5,5};
int b[row2][col2]={1,3,5,6,7,9};
int c[row1][col2];
int *d,*e,*f;
int i,j;
printf("Shared memory in cuda:\n");
printf("\nFirst matrix of size 2*3\n");
for(i=0;i<row1;i++)
{
for(j=0;j<col1;j++)
{
printf("%d\t",a[i][j]);
}
}
printf("\nSecond matrix of size 3*2\n");
for(i=0;i<row2;i++)
{
for(j=0;j<col2;j++)
{
printf("%d\t",b[i][j]);
}
}
hipMalloc((void **)&d,row1*col1*sizeof(int));
hipMalloc((void **)&e,row2*col2*sizeof(int));
hipMalloc((void **)&f,row1*col2*sizeof(int));
hipMemcpy(d,a,row1*col1*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(e,b,row2*col2*sizeof(int),hipMemcpyHostToDevice);
dim3 grid(col2,row1);
matrixproduct<<<grid,col1>>>(d,e,f);
hipMemcpy(c,f,row1*col2*sizeof(int),hipMemcpyDeviceToHost);
printf("\n\nProduct of two matrices:\n ");
for(i=0;i<row1;i++)
{
for(j=0;j<col2;j++)
{
printf("%d\t",c[i][j]);
}
printf("\n");
}
hipFree(d);
hipFree(e);
hipFree(f);
return 0;
}
