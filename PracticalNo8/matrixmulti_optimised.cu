#include "hip/hip_runtime.h"
%%writefile cuda81oti.cu
#include<stdio.h>
#include<hip/hip_runtime.h>
#define row1 2 /* Number of rows of first matrix */
#define col1 3 /* Number of columns of first matrix */
#define row2 3 /* Number of rows of second matrix */
#define col2 2 /* Number of columns of second matrix */
__global__ void matproductsharedmemory(int *l,int *m, int *n)
{
int x=blockIdx.x;
int y=blockIdx.y;
__shared__ int p[col1];
int i;
int k=threadIdx.x;
n[col2*y+x]=0;
p[k]=l[col1*y+k]*m[col2*k+x];
__syncthreads();
for(i=0;i<col1;i++)
n[col2*y+x]=n[col2*y+x]+p[i];
}
int main()
{
int a[row1][col1]={1,2,3,4,5,6};
int b[row2][col2]={1,2,3,4,5,6};
int c[row1][col2];
int *d,*e,*f;
int i,j;
hipMalloc((void **)&d,row1*col1*sizeof(int));
hipMalloc((void **)&e,row2*col2*sizeof(int));
hipMalloc((void **)&f,row1*col2*sizeof(int));
hipMemcpy(d,a,row1*col1*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(e,b,row2*col2*sizeof(int),hipMemcpyHostToDevice);
dim3 grid(col2,row1);
/* Here we are defining two dimensional Grid(collection of blocks) structure. Syntax is
dim3 grid(no. of columns,no. of rows) */
matproductsharedmemory<<<grid,col1>>>(d,e,f);
hipMemcpy(c,f,row1*col2*sizeof(int),hipMemcpyDeviceToHost);
printf("\n Product of two matrices:\n ");
for(i=0;i<row1;i++)
{
for(j=0;j<col2;j++)
{
printf("%d\t",c[i][j]);
}
printf("\n");
}
hipFree(d);
hipFree(e);
hipFree(f);
return 0;
}
