#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void arradd(int *x,int *y, int *z)
{
int id=blockIdx.x;
/* blockIdx.x gives the respective block id which starts from 0 */
z[id]=x[id]+y[id];
}
int main()
{
int a[6];
int b[6];
int c[6];
int *d,*e,*f;
int i;
printf("\n Six elements of first array: ");
for(i=0;i<6;i++)
{
a[i]=i;
printf("%d ",a[i]);
}
printf("\n Six elements of second array: ");
for(i=0;i<6;i++)
{
b[i]=i+1;
printf("%d ",b[i]);
}
hipMalloc((void **)&d,6*sizeof(int));
hipMalloc((void **)&e,6*sizeof(int));
hipMalloc((void **)&f,6*sizeof(int));
hipMemcpy(d,a,6*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(e,b,6*sizeof(int),hipMemcpyHostToDevice);
arradd<<<6,1>>>(d,e,f);
hipMemcpy(c,f,6*sizeof(int),hipMemcpyDeviceToHost);
printf("\nSum of two arrays:\n ");
for(i=0;i<6;i++)
{
printf("%d\t",c[i]);
}
hipFree(d);
hipFree(e);
hipFree(f);
return 0;
}
