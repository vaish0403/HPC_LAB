#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<hip/hip_runtime.h>
#define sc 5


__global__ void arradd(int *x,int *y, int *z)    //kernel definition
{
  int id=blockIdx.x; 
/* blockIdx.x gives the respective block id which starts from 0 */
  z[id]=x[id]+y[id];
}

int main()
{
    int a[6];
    int b[6];
    int c[6];
    int *d,*e,*f;
    int i;
    for(i=0;i<6;i++)
    {
        a[i]=sc;
    }
    printf("Scalar: %d", sc);
	printf("\nVector: ");
        for(i=0;i<6;i++)
        {
           b[i]=i+3;
					    printf("%d ", b[i]);
        }

    hipMalloc((void **)&d,6*sizeof(int));
    hipMalloc((void **)&e,6*sizeof(int));
    hipMalloc((void **)&f,6*sizeof(int));

 hipMemcpy(d,a,6*sizeof(int),hipMemcpyHostToDevice);   
 hipMemcpy(e,b,6*sizeof(int),hipMemcpyHostToDevice);
 
arradd<<<6,1>>>(d,e,f); 

 hipMemcpy(c,f,6*sizeof(int),hipMemcpyDeviceToHost);
    
printf("\nSum :\n ");
    for(i=0;i<6;i++)
    {
        printf("%d\t",c[i]);
    }
    hipFree(d);
    hipFree(e);
    hipFree(f);

    return 0;
}
